#include "hip/hip_runtime.h"
#include "framework.h"

bool nearlyEqual(double a, double b, double epsilon, double relativeTolerance)
{
    double diff = std::fabs(a - b);
    double largest = std::max(std::fabs(a), std::fabs(b));
    return diff <= (epsilon + relativeTolerance * largest);
}

void relu(float *input, int rows, int cols)
{
    for (int i = 0; i < rows; ++i)
    {
        int rowStart = i * cols;
        for (int j = 0; j < cols; ++j)
        {
            input[rowStart + j] = std::max(0.0f, input[rowStart + j]);
        }
    }
}

void softmax(float *input, int rows, int cols)
{
    float *expValues = new float[cols];
    for (int i = 0; i < rows; ++i)
    {
        // Find max value for numerical stability
        float maxVal = input[i * cols];
        for (int j = 1; j < cols; ++j)
        {
            maxVal = std::max(maxVal, input[i * cols + j]);
        }

        // Compute exponentials and sum
        float expSum = 0.0f;

        for (int j = 0; j < cols; ++j)
        {
            int index = i * cols + j;
            // Subtract max for numerical stability
            expValues[j] = std::exp(input[index] - maxVal);
            expSum += expValues[j];
        }

        // Normalize to get probabilities
        for (int j = 0; j < cols; ++j)
        {
            int index = i * cols + j;
            input[index] = expValues[j] / expSum;
        }
    }
    delete[] expValues;
}

void matmul(const float *A, const float *B, float *C, int M, int K, int N)
{
    // Matrix multiplication: C[M x N] = A[M x K] * B[K x N]
    for (int i = 0; i < M; ++i)
        for (int j = 0; j < N; ++j)
        {
            float sum = 0;              // Initialize output
            for (int k = 0; k < K; ++k) // Shared dimension
            {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
}

// assume sum_over_batch
float categorical_crossentropy_loss(uint8_t *y_true, float *y_pred, int batch_size, int num_classes)
{
    float total_loss = 0.0f;

    for (int i = 0; i < batch_size; ++i)
    {
        int true_class = y_true[i];
        float predicted_prob = y_pred[i * num_classes + true_class];

        // Avoid log(0) by clamping probabilities to a small positive value
        const float epsilon = 1e-7f;
        predicted_prob = std::max(predicted_prob, epsilon);
        total_loss -= std::log(predicted_prob);
    }

    return total_loss / batch_size;
}

void categorical_crossentropy_gradient_sparse(const uint8_t *y_true, const float *y_pred, float *d_output, int batch_size, int output_size)
{
    for (int i = 0; i < batch_size; ++i)
    {
        int true_label = y_true[i]; // Sparse label (e.g., 0, 1, 2, ...)
        for (int j = 0; j < output_size; ++j)
        {
            d_output[i * output_size + j] = y_pred[i * output_size + j];
            if (j == true_label)
            {
                d_output[i * output_size + j] -= 1.0f; // Subtract 1 for the true class
            }
        }
    }
}

// int main() {
//     int batch_size = 3;
//     int num_classes = 4;

//     uint8_t y_true[] = {1, 3, 0};  // True class indices for the batch
//     float y_pred[] = {
//         0.1f, 0.7f, 0.1f, 0.1f,
//         0.2f, 0.1f, 0.3f, 0.4f,
//         0.8f, 0.05f, 0.1f, 0.05f
//     };

//     float loss = categorical_crossentropy_loss(y_true, y_pred, batch_size, num_classes);
//     printf("Categorical Crossentropy Loss: %f\n", loss);
//     return 0;
// }
