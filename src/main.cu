#include <stdio.h>
#include <stdint.h>
#include <random>
#include <chrono>
#include <getopt.h>
#include "utils/fashion_mnist.hh"
#include "utils/helpers.hh"
#include "Model/Model.hh"
#include "layer/dense.hh"
#include "layer/relu.hh"
#include "layer/softmax.hh"
#include "loss/categorical_crossentropy.hh"
#include "metrics/accuracy.hh"

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);
    printf("****************************\n");
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

struct HostTimer
{
    std::chrono::high_resolution_clock::time_point start;
    std::chrono::high_resolution_clock::time_point stop;

    // Start the timer
    void Start()
    {
        start = std::chrono::high_resolution_clock::now();
    }

    // Stop the timer
    void Stop()
    {
        stop = std::chrono::high_resolution_clock::now();
    }

    // Get the elapsed time in milliseconds
    float Elapsed()
    {
        std::chrono::duration<float> duration = stop - start;
        return duration.count(); // Returns elapsed time in milliseconds
    }
};


std::mt19937 global_rng(1); // Random number generator
// Model configurations
const int INPUT_SIZE = 784; // Example: MNIST image input size
const int OUTPUT_SIZE = 10;

int main(int argc, char **argv)
{
    printDeviceInfo();
    std::string dataset_path = "";
    int num_epoch = 10;
    int batch_size = 64; // Default value
    float learning_rate = 0.001;
    std::string checkpoint_path = "";
    int opt;

    // Parsing command-line arguments
    while ((opt = getopt(argc, argv, "d:e:b:l:p:")) != -1)
    {
        switch (opt)
        {
        case 'd':
            dataset_path = optarg; // Convert argument to integer
            break;
        case 'e':
            num_epoch = atoi(optarg); // Convert argument to integer
            break;
        case 'b':
            batch_size = atoi(optarg); // Convert argument to integer
            break;
        case 'l':
            learning_rate = atof(optarg); // Convert argument to integer
            break;
        case 'p':
            checkpoint_path = optarg; // Store the checkpoint path
            break;
        default:
            fprintf(stderr, "Usage: %s [-d dataset_path] [-e num_epoch] [-b batchsize] [-l learning_rate] [-p checkpoint_path]\n", argv[0]);
            exit(EXIT_FAILURE);
        }
    }

    const std::string train_imageFilePath = dataset_path + "train-images-idx3-ubyte";
    const std::string train_labelFilePath = dataset_path + "train-labels-idx1-ubyte";
    const std::string test_imageFilePath = dataset_path + "t10k-images-idx3-ubyte";
    const std::string test_labelFilePath = dataset_path + "t10k-labels-idx1-ubyte";


    Layer *layers[] = {
        new Dense(batch_size, INPUT_SIZE, 128, true, global_rng),
        new ReLU(batch_size, 128),
        new Dense(batch_size, 128, 128, true, global_rng),
        new ReLU(batch_size, 128),
        new Dense(batch_size, 128, OUTPUT_SIZE, true, global_rng),
        new Softmax(batch_size, OUTPUT_SIZE)};
    const int NUM_LAYERS = sizeof(layers) / sizeof(layers[0]);

    // Load dataset
    FashionMnist train_set;
    train_set.loadDataset(train_imageFilePath, train_labelFilePath);
    FashionMnist test_set;
    test_set.loadDataset(test_imageFilePath, test_labelFilePath);
    std::cout << "Total train images: " << train_set.getImageCount() << std::endl;
    std::cout << "Total test images: " << test_set.getImageCount() << std::endl;

    int num_batches = train_set.getImageCount() / batch_size;
    float **x_batches = new float *[num_batches];
    uint8_t **y_batches = new uint8_t *[num_batches];
    for (int bi = 0; bi < num_batches; ++bi)
    {
        x_batches[bi] = new float[batch_size * INPUT_SIZE];
        y_batches[bi] = new uint8_t[batch_size];
    }

    int test_num_batches = test_set.getImageCount() / batch_size;
    float **test_x_batches = new float *[test_num_batches];
    uint8_t **test_y_batches = new uint8_t *[test_num_batches];
    for (int bi = 0; bi < test_num_batches; ++bi)
    {
        test_x_batches[bi] = new float[batch_size * INPUT_SIZE];
        test_y_batches[bi] = new uint8_t[batch_size];
    }
    test_set.prepareBatchesWithLabels(batch_size, INPUT_SIZE, test_x_batches, test_y_batches);

    Model model(layers, NUM_LAYERS, batch_size, INPUT_SIZE, OUTPUT_SIZE);
    CategoricalCrossentropy loss_obj(1e-7);
    Accuracy acc_obj;
    float loss_batch;

    std::cout << "\nConfigurations:" << std::endl;
    std::cout << "\tNum epoch: " << num_epoch << std::endl;
    std::cout << "\tBatch size: " << batch_size << std::endl;
    std::cout << "\tLearning rate: " << learning_rate << std::endl;
    std::cout << "\tCheckpoint: " << checkpoint_path << std::endl;


    HostTimer epoch_timer;
    HostTimer total_timer;
    total_timer.Start();

    // tmp malloc
    float *tmp_y_pred = new float[batch_size * OUTPUT_SIZE];

    for (int epoch = 0; epoch < num_epoch; epoch++)
    {
        // Start timing
        epoch_timer.Start();

        printf("====================Epoch (%d/%d)====================\n", epoch + 1, num_epoch);
        // reshuffle train after each epoch
        train_set.shuffle(global_rng);
        train_set.prepareBatchesWithLabels(batch_size, INPUT_SIZE, x_batches, y_batches);
        loss_obj.reset_state();
        acc_obj.reset_state();
        for (int bi = 0; bi < num_batches; ++bi)
        {
            model.forward(x_batches[bi], tmp_y_pred);

            loss_batch = loss_obj.forward(y_batches[bi], tmp_y_pred, batch_size, OUTPUT_SIZE);
            loss_obj.update_state(loss_batch);
            acc_obj.update_state(tmp_y_pred, y_batches[bi], batch_size, OUTPUT_SIZE);

            model.backward(y_batches[bi], tmp_y_pred, &loss_obj);
            model.update_weights(learning_rate);

            if (bi % 100 == 0 || bi == num_batches - 1)
            {
                printf("Iter (%d/%d): loss - %f, acc - %f\n",
                       bi, num_batches - 1,
                       loss_obj.compute_average_loss(), acc_obj.compute());
            }
        }
        loss_obj.reset_state();
        acc_obj.reset_state();

        for (int bi = 0; bi < test_num_batches; ++bi)
        {
            model.forward(test_x_batches[bi], tmp_y_pred);
            loss_batch = loss_obj.forward(test_y_batches[bi], tmp_y_pred, batch_size, OUTPUT_SIZE);
            loss_obj.update_state(loss_batch);
            acc_obj.update_state(tmp_y_pred, test_y_batches[bi], batch_size, OUTPUT_SIZE);
        }
        printf("Validation: loss - %f, acc - %f\n", loss_obj.compute_average_loss(), acc_obj.compute());

        // Stop timing
        epoch_timer.Stop();

        // Get and print the elapsed time
        printf("Epoch time: %f seconds\n", epoch_timer.Elapsed());
    }
    total_timer.Stop();
    printf("Total time: %f seconds\n", total_timer.Elapsed());

    if (checkpoint_path != "")
        model.save_weights(checkpoint_path);

    // DONT DELETE COMMENTED CODE BELOW  (for verify)

    // Layer *layers2[] = {
    // new Dense(batch_size, INPUT_SIZE, 128, global_rng),
    // new ReLU(batch_size, 128),
    // new Dense(batch_size, 128, 128, global_rng),
    // new ReLU(batch_size, 128),
    // new Dense(batch_size, 128, OUTPUT_SIZE, global_rng),
    // new Softmax(batch_size, OUTPUT_SIZE)};
    // Model model2(layers2, NUM_LAYERS, batch_size, INPUT_SIZE, OUTPUT_SIZE);
    // model2.load_weights("weight_ne.bin");
    // float* tmp_batch = new float[batch_size * OUTPUT_SIZE];
    // model2.forward(test_x_batches[0], tmp_batch);
    // std::cout << "Ori preds" << std::endl;
    // for (int ii = 0; ii < batch_size; ii++) {
    //     for (int jj = 0; jj < OUTPUT_SIZE; jj++) {
    //         std::cout << test_y_pred_batches[0][ii * OUTPUT_SIZE + jj] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << "Loaded preds" << std::endl;
    // for (int ii = 0; ii < batch_size; ii++) {
    //     for (int jj = 0; jj < OUTPUT_SIZE; jj++) {
    //         std::cout << tmp_batch[ii * OUTPUT_SIZE + jj] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // delete[] tmp_batch;



    // Deallocate
    delete[] tmp_y_pred;
    for (int i = 0; i < num_batches; ++i)
    {
        delete[] x_batches[i];
        delete[] y_batches[i];
    }
    delete[] x_batches;
    delete[] y_batches;

    for (int i = 0; i < test_num_batches; ++i)
    {
        delete[] test_x_batches[i];
        delete[] test_y_batches[i];
    }
    delete[] test_x_batches;
    delete[] test_y_batches;
    return 0;
}
